#include "hip/hip_runtime.h"
// module load CUDA/10.1.243-GCC-8.3.0
// nvcc -Xcompiler -o SparseMV SparseMV.cu mtx_sparse.c
// srun --reservation=fri --gpus=1 SparseMV data/scircuit.mtx
// srun --reservation=fri --gpus=1 SparseMV data/pdb1HYS.mtx
// srun --reservation=fri -G1 -n1 sparseMV data/pwtk.mtx

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "mtx_sparse.h"

#define THREADS_PER_BLOCK 64
#define THREADS_PER_ROW 16
#define REPEAT 1

/* README for tests done on matrix pwtk:
THREADS_PER_BLOCK: 256
8: Times: 14.6 ms(COO_cpu), 16.2 ms(CSR_gpu), 16.1 ms (CSRPar_gpu), 19.3 ms(ELL_gpu)
16: Times: 14.7 ms(COO_cpu), 16.6 ms(CSR_gpu), 16.5 ms (CSRPar_gpu), 33.0 ms(ELL_gpu)
32: Times: 15.7 ms(COO_cpu), 17.7 ms(CSR_gpu), 17.6 ms (CSRPar_gpu), 62.2 ms(ELL_gpu)
64: Times: 14.6 ms(COO_cpu), 20.0 ms(CSR_gpu), 19.9 ms (CSRPar_gpu), 121.6 ms(ELL_gpu)
128: Times: 14.6 ms(COO_cpu), 24.7 ms(CSR_gpu), 24.6 ms (CSRPar_gpu), 240.0 ms(ELL_gpu)

THREADS_PER_BLOCK: 128
8: Times: 14.6 ms(COO_cpu), 8.4 ms(CSR_gpu), 8.3 ms (CSRPar_gpu), 16.6 ms(ELL_gpu)
16: Times: 14.6 ms(COO_cpu), 9.0 ms(CSR_gpu), 8.8 ms (CSRPar_gpu), 31.1 ms(ELL_gpu)
32: Times: 14.7 ms(COO_cpu), 10.1 ms(CSR_gpu), 10.0 ms (CSRPar_gpu), 61.0 ms(ELL_gpu)
64: Times: 14.7 ms(COO_cpu), 12.4 ms(CSR_gpu), 12.2 ms (CSRPar_gpu), 119.6 ms(ELL_gpu)

THREADS_PER_BLOCK: 64
8: Times: 14.6 ms(COO_cpu), 9.1 ms(CSR_gpu), 9.0 ms (CSRPar_gpu), 118.7 ms(ELL_gpu)
16: Times: 14.6 ms(COO_cpu), 5.2 ms(CSR_gpu), 5.1 ms (CSRPar_gpu), 30.4 ms(ELL_gpu)
32: Times: 14.6 ms(COO_cpu), 6.5 ms(CSR_gpu), 6.3 ms (CSRPar_gpu), 59.8 ms(ELL_gpu)
64: Times: 14.6 ms(COO_cpu), 9.1 ms(CSR_gpu), 9.0 ms (CSRPar_gpu), 118.7 ms(ELL_gpu)

THREADS_PER_BLOCK: 512
8: Times: 14.6 ms(COO_cpu), 16.2 ms(CSR_gpu), 16.3 ms (CSRPar_gpu), 19.3 ms(ELL_gpu)
16: Times: 14.6 ms(COO_cpu), 32.2 ms(CSR_gpu), 32.0 ms (CSRPar_gpu), 38.3 ms(ELL_gpu)
32: Times: 14.7 ms(COO_cpu), 33.0 ms(CSR_gpu), 33.0 ms (CSRPar_gpu), 65.7 ms(ELL_gpu)
64: Times: 14.7 ms(COO_cpu), 35.2 ms(CSR_gpu), 35.1 ms (CSRPar_gpu), 124.3 ms(ELL_gpu)


We can conclude the testing here and we get out fastest result for the parrallel algorith with 64 threads per block and 16 threads per row.
The speed up between ELL_gpu and CSRPar_gpu is also interesting for different values.
*/

__global__ void mCSRxVecPar(int *rowptr, int *col, float *data, float *vin, float *vout, int rows)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    if (gid < rows)
    {
        float sum = 0.0f;
        int row_start = rowptr[gid];
        int row_end = rowptr[gid + 1];

        for (int j = row_start; j < row_end; j++)
            sum += data[j] * vin[col[j]];

        vout[gid] = sum;
    }
}

__global__ void mCSRxVec(int *rowptr, int *col, float *data, float *vin, float *vout, int rows)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    if (gid < rows)
    {
        float sum = 0.0f;
        for (int j = rowptr[gid]; j < rowptr[gid + 1]; j++)
            sum += data[j] * vin[col[j]];
        vout[gid] = sum;
    }
}

__global__ void mELLxVec(int *col, float *data, float *vin, float *vout, int rows, int elemsinrow)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    if (gid < rows)
    {
        float sum = 0.0f;
        int idx;
        for (int j = 0; j < elemsinrow; j++)
        {
            idx = j * rows + gid;
            sum += data[idx] * vin[col[idx]];
        }
        vout[gid] = sum;
    }
}

int main(int argc, char *argv[])
{
    FILE *f;
    struct mtx_COO h_mCOO;
    struct mtx_CSR h_mCSR;
    struct mtx_ELL h_mELL;
    int repeat;

    if (argc < 2)
    {
        fprintf(stderr, "Usage: %s [martix-market-filename]\n", argv[0]);
        exit(1);
    }
    else
    {
        if ((f = fopen(argv[1], "r")) == NULL)
            exit(1);
    }

    // create sparse matrices
    if (mtx_COO_create_from_file(&h_mCOO, f) != 0)
        exit(1);
    mtx_CSR_create_from_mtx_COO(&h_mCSR, &h_mCOO);
    mtx_ELL_create_from_mtx_CSR(&h_mELL, &h_mCSR);

    // allocate vectors
    float *h_vecIn = (float *)malloc(h_mCOO.num_cols * sizeof(float));
    for (int i = 0; i < h_mCOO.num_cols; i++)
        h_vecIn[i] = 1.0;
    float *h_vecOutCOO_cpu = (float *)calloc(h_mCOO.num_rows, sizeof(float));
    float *h_vecOutCSR_gpu = (float *)calloc(h_mCSR.num_rows, sizeof(float));
    float *h_vecOutELL_gpu = (float *)calloc(h_mELL.num_rows, sizeof(float));
    float *h_vecOutCSRpar = (float *)calloc(h_mCSR.num_rows, sizeof(float));

    // compute with COO
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (repeat = 0; repeat < REPEAT; repeat++)
    {
        for (int i = 0; i < h_mCOO.num_rows; i++)
            h_vecOutCOO_cpu[i] = 0.0;
        for (int i = 0; i < h_mCOO.num_nonzeros; i++)
            h_vecOutCOO_cpu[h_mCOO.row[i]] += h_mCOO.data[i] * h_vecIn[h_mCOO.col[i]];
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float dtimeCOO_cpu = 0;
    hipEventElapsedTime(&dtimeCOO_cpu, start, stop);

    // allocate memory on device and transfer data from host
    // CSR
    int *d_mCSRrowptr, *d_mCSRcol;
    float *d_mCSRdata;
    hipMalloc((void **)&d_mCSRrowptr, (h_mCSR.num_rows + 1) * sizeof(int));
    hipMalloc((void **)&d_mCSRcol, (h_mCSR.num_nonzeros + 1) * sizeof(int));
    hipMalloc((void **)&d_mCSRdata, h_mCSR.num_nonzeros * sizeof(float));
    hipMemcpy(d_mCSRrowptr, h_mCSR.rowptr, (h_mCSR.num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mCSRcol, h_mCSR.col, h_mCSR.num_nonzeros * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mCSRdata, h_mCSR.data, h_mCSR.num_nonzeros * sizeof(float), hipMemcpyHostToDevice);
    // ELL
    int *d_mELLcol;
    float *d_mELLdata;
    hipMalloc((void **)&d_mELLcol, h_mELL.num_elements * sizeof(int));
    hipMalloc((void **)&d_mELLdata, h_mELL.num_elements * sizeof(float));
    hipMemcpy(d_mELLcol, h_mELL.col, h_mELL.num_elements * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mELLdata, h_mELL.data, h_mELL.num_elements * sizeof(float), hipMemcpyHostToDevice);

    // vectors
    float *d_vecIn, *d_vecOut;
    hipMalloc((void **)&d_vecIn, h_mCOO.num_cols * sizeof(float));
    hipMalloc((void **)&d_vecOut, h_mCOO.num_rows * sizeof(float));
    hipMemcpy(d_vecIn, h_vecIn, h_mCSR.num_cols * sizeof(float), hipMemcpyHostToDevice);

    // Divide work
    dim3 blocksize(THREADS_PER_BLOCK / THREADS_PER_ROW, THREADS_PER_ROW);

    // CSR
    dim3 gridsize_CSR((h_mCSR.num_rows - 1) / blocksize.x + 1);

    // CSRPar
    // TODO: Fix me
    dim3 gridsize_CSRpar((h_mCSR.num_rows - 1) / blocksize.x + 1);

    // ELL
    dim3 gridsize_ELL((h_mELL.num_rows - 1) / blocksize.x + 1);

    // CSR execute
    hipEventRecord(start);
    for (repeat = 0; repeat < REPEAT; repeat++)
    {
        mCSRxVec<<<gridsize_CSR, blocksize>>>(d_mCSRrowptr, d_mCSRcol, d_mCSRdata, d_vecIn, d_vecOut, h_mCSR.num_rows);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float dtimeCSR_gpu = 0;
    hipEventElapsedTime(&dtimeCSR_gpu, start, stop);
    hipMemcpy(h_vecOutCSR_gpu, d_vecOut, h_mCSR.num_rows * sizeof(float), hipMemcpyDeviceToHost);

    // CSRPar execute
    hipEventRecord(start);
    for (repeat = 0; repeat < REPEAT; repeat++)
    {
        mCSRxVecPar<<<gridsize_CSRpar, blocksize>>>(d_mCSRrowptr, d_mCSRcol, d_mCSRdata, d_vecIn, d_vecOut, h_mCSR.num_rows);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float dtimeCSRPar_gpu = 0;
    hipEventElapsedTime(&dtimeCSRPar_gpu, start, stop);
    hipMemcpy(h_vecOutCSRpar, d_vecOut, h_mCSR.num_rows * sizeof(float), hipMemcpyDeviceToHost);

    // ELL write, execute, read
    hipEventRecord(start);
    for (repeat = 0; repeat < REPEAT; repeat++)
    {
        mELLxVec<<<gridsize_ELL, blocksize>>>(d_mELLcol, d_mELLdata, d_vecIn, d_vecOut, h_mELL.num_rows, h_mELL.num_elementsinrow);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float dtimeELL_gpu = 0;
    hipEventElapsedTime(&dtimeELL_gpu, start, stop);
    hipMemcpy(h_vecOutELL_gpu, d_vecOut, h_mELL.num_rows * sizeof(float), hipMemcpyDeviceToHost);

    // release device memory
    hipFree(d_mCSRrowptr);
    hipFree(d_mCSRcol);
    hipFree(d_mCSRdata);
    hipFree(d_mELLcol);
    hipFree(d_mELLdata);
    hipFree(d_vecIn);
    hipFree(d_vecOut);

    // output
    printf("Matrix: %s, size: %d x %d, nonzero: %d, max elems in row: %d\n", argv[1], h_mCOO.num_rows, h_mCOO.num_cols, h_mCOO.num_nonzeros, h_mELL.num_elementsinrow);
    int errorsCSR_gpu = 0;
    int errorsCSRPar_gpu = 0;
    int errorsELL_gpu = 0;
    for (int i = 0; i < h_mCOO.num_rows; i++)
    {

        if (fabs(h_vecOutCOO_cpu[i] - h_vecOutCSR_gpu[i]) > 1e-4)
            errorsCSR_gpu++;
        if (fabs(h_vecOutCOO_cpu[i] - h_vecOutCSRpar[i]) > 1e-4)
            errorsCSRPar_gpu++;
        if (fabs(h_vecOutCOO_cpu[i] - h_vecOutELL_gpu[i]) > 1e-4)
            errorsELL_gpu++;
    }
    printf("Errors: %d(CSR_gpu), %d(CSRPar_gpu), %d(ELL_gpu)\n", errorsCSR_gpu, errorsCSRPar_gpu, errorsELL_gpu);
    printf("Times: %.1f ms(COO_cpu), %.1f ms(CSR_gpu), %.1f ms (CSRPar_gpu), %.1f ms(ELL_gpu)\n\n", dtimeCOO_cpu, dtimeCSR_gpu, dtimeCSRPar_gpu, dtimeELL_gpu);
    // release host memory
    mtx_COO_free(&h_mCOO);
    mtx_CSR_free(&h_mCSR);
    mtx_ELL_free(&h_mELL);

    free(h_vecIn);
    free(h_vecOutCOO_cpu);
    free(h_vecOutCSR_gpu);
    free(h_vecOutELL_gpu);

    return 0;
}
